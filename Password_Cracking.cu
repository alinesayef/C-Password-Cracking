#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__device__ char pwdFound[4];

__device__ __host__ void CudaCrypt(char* rawPassword, char* encryptedPassword) {
	encryptedPassword[0] = rawPassword[0] + 2;
	encryptedPassword[1] = rawPassword[0] - 2;
	encryptedPassword[2] = rawPassword[0] + 1;
	encryptedPassword[3] = rawPassword[1] + 3;
	encryptedPassword[4] = rawPassword[1] - 3;
	encryptedPassword[5] = rawPassword[1] - 1;
	encryptedPassword[6] = rawPassword[2] + 2;
	encryptedPassword[7] = rawPassword[2] - 2;
	encryptedPassword[8] = rawPassword[3] + 4;
	encryptedPassword[9] = rawPassword[3] - 4;
	encryptedPassword[10] = '\0';

	for (int i = 0; i < 10; i++) {
		if (i >= 0 && i < 6) { //checking all lower case letter limits
			if (encryptedPassword[i] > 122) {
				encryptedPassword[i] = (encryptedPassword[i] - 122) + 97;
			}
			else if (encryptedPassword[i] < 97) {
				encryptedPassword[i] = (97 - encryptedPassword[i]) + 97;
			}
		}
		else { //checking number section
			if (encryptedPassword[i] > 57) {
				encryptedPassword[i] = (encryptedPassword[i] - 57) + 48;
			}
			else if (encryptedPassword[i] < 48) {
				encryptedPassword[i] = (48 - encryptedPassword[i]) + 48;
			}
		}
	}
}

__global__ void crack(char* alphabet, char* numbers, char* encryptedPwdToFind) {
	char genRawPass[4];
	
	genRawPass[0] = alphabet[blockIdx.x];
	genRawPass[1] = alphabet[blockIdx.y];
	genRawPass[2] = numbers[threadIdx.x];
	genRawPass[3] = numbers[threadIdx.y];
	
	char newPassword[11];


	CudaCrypt(genRawPass, newPassword); //run the CudaCrypt function


	for (int i = 0; i < 10; ++i) {
		if (newPassword[i] != encryptedPwdToFind[i]) {
			return;
		}
	}
	
	pwdFound[0] = genRawPass[0];
	pwdFound[1] = genRawPass[1];
	pwdFound[2] = genRawPass[2];
	pwdFound[3] = genRawPass[3];
	
	printf("Encrypted Password provided: %s\n", encryptedPwdToFind);
	printf("Password found: %c%c%c%c\n", pwdFound[0],pwdFound[1],pwdFound[2],pwdFound[3]); //print the found password
}

int main(int argc, char** argv) {
	char* commands = "Usage:\nhash [clear password]\ndecrypt [hash of the password]\nhashndecrypt [clear password]\n\n"; //sets possible commands to a "command" variable.

	char encryptedPwdToFind[11];
	strcpy(encryptedPwdToFind, "cxbdwy2745"); // The default encrypted password to decrypt.

//Determine the type of command input
	if (argc == 3) {
		if (!strcmp(argv[1], "hash")) {
			if (strlen(argv[2]) != 4) {
				printf("%s\n", commands);
				return 1;
			}
			char hash[11];
			CudaCrypt(argv[2], hash);
			printf("Hash generated: %s\n", hash);
			return 0;
		}
		else if (!strcmp(argv[1], "decrypt")) {
			if (strlen(argv[2]) != 10) {
				printf("%s\n", commands);
				return 1;
			}
			strcpy(encryptedPwdToFind, argv[2]);
		}
		else if (!strcmp(argv[1], "hashndecrypt")) {
			if (strlen(argv[2]) != 4) {
				printf("%s\n", commands);
				return 1;
			}
			CudaCrypt(argv[2], encryptedPwdToFind);
		}
		else {
			printf("%s\n", commands);
			return 1;
		}
	}
	
	//Output possible commands to the user
	else if (argc > 1) {
		printf("%s\n", commands);
		return 1;
	}

	printf("Trying to find the password for %s\n", encryptedPwdToFind);

//Array for numbers and letters of the alphabet
	char cpuAlphabet[26] = { 'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z' };
	char cpuNumbers[10] = { '0','1','2','3','4','5','6','7','8','9' };

//Allocate memory for the letters of the alphabet and copy them from host to the device
	char* gpuAlphabet;
	hipMalloc((void**)&gpuAlphabet, sizeof(char) * 26);
	hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);

//Allocate memory for the numbers and copy them from host to the device
	char* gpuNumbers;
	hipMalloc((void**)&gpuNumbers, sizeof(char) * 10);
	hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 10, hipMemcpyHostToDevice);


	char* gpuEncryptedPwdToFind;
	hipMalloc((void**)&gpuEncryptedPwdToFind, sizeof(char) * 11);
	hipMemcpy(gpuEncryptedPwdToFind, encryptedPwdToFind, sizeof(char) * 11, hipMemcpyHostToDevice);

	crack << < dim3(26, 26), dim3(10, 10) >> > (gpuAlphabet, gpuNumbers, gpuEncryptedPwdToFind);

	hipDeviceSynchronize();
	
	//Free allocated memory
	hipFree(gpuAlphabet);
	hipFree(gpuNumbers);
	hipFree(gpuEncryptedPwdToFind);
	
}













